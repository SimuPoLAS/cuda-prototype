
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// TODO: find proper alignment (__align__(x))
struct node {
    int value;
    int children[8];
    int next;
};

// function to add the elements of a static array and a more dynamic tree structure
__global__
void add(int n, float* a, node* nodes, float* b) {
    printf("hello world from the device!\n"); 
    int index = threadIdx.x;
    int stride = blockDim.x;
    int childID = 0;
    //int childID = nodes[0].children[0];
    for (int i = index; i < n; i += stride)
        b[i] += a[i] + nodes[childID].value;
}

int main(void) {
    // 1 + 8 + 8*8 + 8*8*8
    int N = 585;
    //int N = 1<<20;

    // create buffers
    float *a, *b;

    // initialize a and b arrays on the host
    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // create node buffer
    node nodes[N];
    // initialize "tree" structure
    // calculate the required depth for our structure first
    int depth = 1;
    int x = N;
    while (x >= 8) {
        x -= x % 8;
        x /= 8;
        depth++;
    }

    int rootnodelength = 1;
    for (int d = 0; d < depth; d++) {
        rootnodelength *= 8;
    }
    // pad rootnodelength with unused fields
    rootnodelength += N % rootnodelength;
    rootnodelength += 1;

    for (int i = 0; i < N; i++) {
        // replace depth with current depth
        nodes[i].value = (100 * depth) + i;
        // logarithm of x base b = log(x)/log(b)
        // TODO: create some sort of magical function to get the current depth for i
        // using a log base 8 function and more sik math skills
        // then populate nodes and their indeces in accordance with the current depth

        // for testing purposes **ONLY**
        for (int j = 0; j < 8; j++) {
            // set all children to node number 1 (yeah I know pretty lame)
            nodes[i].children[j] = 1;
        }
        // also set next temporarily
        nodes[i].next = 2;
    }

    printf("aaaaaaaaaaaaaaaa\n");
    // create space in memory for a copy of nodes to be used by the device
    node *nodes_d;
    hipMallocManaged(&a, N * sizeof(float));
    hipMallocManaged(&b, N * sizeof(float));
    hipMallocManaged((void**) &nodes_d, sizeof(node) * sizeof(nodes));

    // copy nodes buffer to nodes_d (memory space we allocated for the device)
    hipMemcpy(nodes_d, nodes, sizeof(node) * sizeof(node), hipMemcpyHostToDevice);
    // suffering
    printf("AAAAAAAAAAAA\n");
    add<<<1, 256>>>(N, a, nodes, b);

    // wait for gpu (blocks thread till end is signaled)
    hipDeviceSynchronize();
    // even more suffering
    printf("aaaaaabbbbbbbbbbbbbbbb");

    /*
    float max_error = 0.0f;
    for (int i = 0; i < N; i++)
        max_error = fmax(max_error, fabs(b[i]));
    std::cout << "max error: " << max_error << std::endl;
    */

    float median = 0.0f;
    for (int i = 0; i < N; i++) {
        median = fmax(median, fabs(b[i]));
    }
    std::cout << "median : " << median << "\n";

    // free memory
    hipFree(a);
    hipFree(b);
    hipFree(nodes);

    return 0;
}
